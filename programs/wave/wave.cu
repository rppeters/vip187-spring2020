#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <gd.h>
#include <assert.h>
#include <time.h>


//array index to km
#define XIND2KM(x) ((x)*(dimx/nx))
#define YIND2KM(y) ((y)*(dimy/ny))

#define FRAMEDELAY 0
#define PHEIGHT 1
#define PWIDTH 1
#define MAXCOLORS 256
#define PRESSURERANGE 2
#define MAXBLOCKS 512
#define MAXTHREAD 512

//globals
double c = 1.5;		//speed of wave (km/s)
double lambda = 3; 	//wave length (km)
double sigma = 4;	//width of gaussian disturbance

int dimx = 100; //metric distance of x in km
int dimy = 100; //metric distance of y in km

int nx, ny, sx, sy; 
int timesteps = 5;
double *p_arr;
int radius = 50;

FILE *giffile;	
gdImagePtr im, previm;
int *colors;
int framecount = 0;
double *buf;

int fps = 10;
int maxdist;

int nthreads = 512;

__device__ int d_nx, d_ny, d_sx, d_sy, d_dimx, d_dimy;
__device__ double d_c, d_lambda, d_sigma;

__global__ void calculateWaveProp(double *arr, double time) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	double range = sqrt(pow((x-d_sx)*d_dimx/ d_nx, 2) + pow((y-d_sy)*d_dimy/d_ny, 2)); 				
	

	//gaussian pulse generation based on range and time
	double p0 = exp(-0.5 * pow((((d_c * time)-range) / d_sigma ),2));
	double p = p0 * cos(2 * M_PI * ((d_c * time)-range)/ d_lambda);			

	//printf("p[%d][%d]=%3.3f, range=%3.3f, p0=%3.3f,xdim=%d,ydim=%d\n",x,y,p,sqrt(pow(x-d_sx,2) + pow(y-d_sy,2)),p0,d_dimx,d_dimy);
	arr[x*d_nx + y] = p;
}



void printPressureArray();

void write_frame(double *p_arr, double time) {	
	im = gdImageCreate(nx*PWIDTH,ny*PHEIGHT);
	if (time == 0) {
		colors = (int *)malloc(MAXCOLORS*sizeof(int));
		for (int j = 0; j < MAXCOLORS; j++) {
			colors[j] = gdImageColorAllocate(im, j, 0, MAXCOLORS-j-1);
		}
		gdImageGifAnimBegin(im,giffile,1,-1);
	} else {
		gdImagePaletteCopy(im, previm);
	}
	
	for (int i = 0; i < nx; i++) {
		for (int j = 0; j < ny; j++) {
			int color = (int)(((1+p_arr[i*nx + j])*MAXCOLORS)/PRESSURERANGE);

			assert(color >= 0);
			if (color >= MAXCOLORS) color = MAXCOLORS-1;
			
			gdImageFilledRectangle(im, i*PWIDTH, j*PHEIGHT, (i+1)*PWIDTH-1, (j+1)*PHEIGHT-1, colors[color]);
		}

	}

	if (time == 0) {
		//use a large frame delay to give buffer time for eog to open .gif file
		gdImageGifAnimAdd(im,giffile, 0, 0, 0, 200, gdDisposalNone, NULL);	
	} else {	
		gdImageSetPixel(im, 0, 0, framecount%2);
		gdImageGifAnimAdd(im, giffile, 0, 0, 0, FRAMEDELAY , gdDisposalNone, previm);
		gdImageDestroy(previm);
	}

	previm = im;
	im = NULL;
	framecount++;

#ifdef DEBUG
	if (framecount < 10) printPressureArray();
#endif	
}

void printArray(double* arr) {
	for (int i = 0; i < nx; i++) {
		for (int j = 0; j < ny; j++) {
			printf("%6.2f ",arr[i*nx + j]);
		}
		printf("\n");
	}
	printf("\n");
}

void init(int argc, char *argv[]) {
	nx = atoi(argv[1]);
	ny = atoi(argv[2]);
	sx = atoi(argv[3]);
	sy = atoi(argv[4]);
	timesteps = atoi(argv[5]);

	maxdist = atoi(argv[6]);
	if (nx >=  ny) {
		dimx = maxdist; 
		dimy = ny*maxdist/nx;
	} else {
		dimy = maxdist;
		dimx = nx*maxdist/ny;
	}

	long int totalPoints = nx*ny*timesteps*fps;
	printf("Total Points: %ld\n",totalPoints);

	//ignore warnings for gif opening size if gif shall not be opened
	char *filename = argv[7];
	if (filename != "DONOTOPEN.gif") assert(totalPoints < 500000000);

	giffile = fopen(argv[7],"wb");
}

void calculateEvenSquareDistribution(int *nxb, int *nyb) {
	int xb = (int)(sqrt(nx));
	while (xb > 0) {
		if (nx % xb == 0) break;
		xb--;
	}

	int yb = (int)(sqrt(ny));
	while (yb > 0) {
		if (ny % yb == 0) break;
		yb--;
	}
	*nxb = xb; *nyb = yb;
}

void calculateEvenMaxDistribution(int *nxb, int *nyb) {
	int xb = MAXBLOCKS;
	while (xb > 0) {
		if (nx % xb == 0 && xb < MAXBLOCKS) break;
		xb--;
	}

	int yb = MAXBLOCKS;
	while (yb > 0) {
		if (ny % yb == 0 && yb < MAXBLOCKS) break;
		yb--;
	}
	*nxb = xb; *nyb = yb;
}

void calculateMaxCacheHitsDistribution(int *nxb, int *nyb) {
	//maximize cache  (ie blocks are 1 row but split maximally)
	*nxb = nx;
	int yb;
	if (MAXBLOCKS > nx) {
		yb = (int) (sqrt(nx));
	} else {yb = nx;}
	
	while (yb > 0) {
		if (ny % yb == 0 && yb < MAXBLOCKS) break;
		yb--;
	}
	*nyb = yb;
}

int main(int argc, char *argv[]) {
	init(argc, argv);
	double *d_p_arr, *p_arr;		
	//allocate array on CPU 
	p_arr = (double *)malloc(ny*nx*sizeof(double));	

	//allocate and copy to GPU
	hipMalloc((void**)&d_p_arr, nx*ny*sizeof(double));
	

	//copy CPU globals to GPU variables
	hipMemcpyToSymbol(HIP_SYMBOL(d_c), &c, sizeof(double), 0,hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_nx), &nx, sizeof(int), 0,hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_ny), &ny, sizeof(int), 0,hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_sx), &sx, sizeof(int), 0,hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_sy), &sy, sizeof(int), 0,hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_dimx), &dimx, sizeof(int), 0,hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_dimy), &dimy, sizeof(int), 0,hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_lambda), &lambda, sizeof(double),0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_sigma), &sigma, sizeof(double), 0,hipMemcpyHostToDevice);
	
	//run wave propagation
	double timeinc = (double)(1.0/fps); //time increments

	int nxblocks = 0;
	int nyblocks = 0;

	//determine which block distribution scheme to use 
	int dist_type = atoi(argv[8]);
	if (dist_type == 0) calculateEvenSquareDistribution(&nxblocks,&nyblocks);
	else if (dist_type == 1) calculateEvenMaxDistribution(&nxblocks,&nyblocks);
	else if (dist_type == 2) calculateMaxCacheHitsDistribution(&nxblocks, &nyblocks);
	else assert(0);
	
	dim3 numBlocks(nxblocks,nyblocks,1);
	dim3 numThreadsPerBlock(nx/nxblocks,ny/nyblocks,1);

	clock_t start,end;
	start = clock();
	for (double t = 0; t < timesteps; t += timeinc) { 	
		calculateWaveProp<<<numBlocks,numThreadsPerBlock>>>(d_p_arr, t);
		hipDeviceSynchronize();

		hipMemcpy(p_arr, d_p_arr, nx*ny*sizeof(double),hipMemcpyDeviceToHost);
		
		write_frame(p_arr, t);
	}

	
	//print timings and results
	end = clock();
	printf("Time: %3.5f\n",((double)(end-start)/CLOCKS_PER_SEC));

	printf("Gif of size %dx%d created:\n\tNumber of Frames: %d \n\tFPS: %d \n\tTotal Time: %d\n",nx,ny,timesteps*fps,fps,timesteps);
	printf("\tLength of X Dim: %d km\n\tLength of Y Dim: %d km\n", dimx, dimy);
	printf("\tX-Dim Blocks: %d\n\tY-Dim Blocks: %d\n\tX-Dim of Threads: %d\n\tY-Dim of Threads: %d\n\n\n",nxblocks, nyblocks, nx/nxblocks, ny/nyblocks);
	
	
	//free
	hipFree(d_p_arr);
	gdImageGifAnimEnd(giffile);
	fclose(giffile);

	free(p_arr);
	free(colors);
	gdImageDestroy(previm);

	return 0;
}
